#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "device_functions.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <assert.h>
#include <windows.h>
#include <intrin.h>
#else
/* compile with: nvcc -O3 -maxrregcount=32 hw2.cu -o hw2 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <assert.h>
#include <string.h>
#endif

#define IMG_DIMENSION 32
#define N_IMG_PAIRS 10000
#define NREQUESTS 10000
#define NSTREAMS 64
#define MAXREGCOUNT 32
#define QUEUENODES 10

typedef unsigned char uchar;
#define OUT

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define CUDA_CHECK(f) do {                                                                  \
    cudaError_t e = f;                                                                      \
    if (e != cudaSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, cudaGetErrorString(e));    \
        return 1;                                                                           \
    }                                                                                       \
} while (0)
#else
#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)
#endif

#define SQR(a) ((a) * (a))

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#if !defined(WIN32_LEAN_AND_MEAN)
#define WIN32_LEAN_AND_MEAN
#endif
#define __sync_synchronize() _ReadWriteBarrier()
double static inline get_time_msec(void) {
    LARGE_INTEGER t;
    static double oofreq;
    static int checkedForHighResTimer;
    static BOOL hasHighResTimer;

    if (!checkedForHighResTimer)
    {
        hasHighResTimer = QueryPerformanceFrequency(&t);
        oofreq = 1000.0 / (double)t.QuadPart;
        checkedForHighResTimer = 1;
    }

    if (hasHighResTimer)
    {
        QueryPerformanceCounter(&t);
        return (double)t.QuadPart * oofreq;
    }
    else
    {
        return (double)GetTickCount();
    }
}
void usleep(unsigned int usec)
{
    HANDLE timer;
    LARGE_INTEGER ft;

    ft.QuadPart = -(10 * (__int64)usec);

    timer = CreateWaitableTimer(NULL, TRUE, NULL);
    SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
    WaitForSingleObject(timer, INFINITE);
    CloseHandle(timer);
}
int rand_r(unsigned int *pseed) {
    srand(*pseed);
    return rand();
}
#else
double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}
#endif
struct stream_node {
    hipStream_t Stream;
    int stream_id;
    int req_in_processing;
};
typedef stream_node streamNode;

typedef struct _thread_node {
    volatile int req_id;
    volatile double result;
} thread_node;

typedef struct _threads_queue {
    volatile uchar read_index;
    volatile uchar write_index;
    volatile thread_node queue_array[QUEUENODES];
} threads_queue;

/* we'll use these to rate limit the request load */
struct rate_limit_t {
    double last_checked;
    double lambda;
    unsigned seed;
};

void rate_limit_init(struct rate_limit_t *rate_limit, double lambda, int seed) {
    rate_limit->lambda = lambda;
    rate_limit->seed = (seed == -1) ? 0 : seed;
    rate_limit->last_checked = 0;
}

int rate_limit_can_send(struct rate_limit_t *rate_limit) {
    if (rate_limit->lambda == 0) return 1;
    double now = get_time_msec() * 1e-3;
    double dt = now - rate_limit->last_checked;
    double p = dt * rate_limit->lambda;
    rate_limit->last_checked = now;
    if (p > 1) p = 1;
    double r = (double)rand_r(&rate_limit->seed) / RAND_MAX;
    return (p > r);
}

void rate_limit_wait(struct rate_limit_t *rate_limit) {
    while (!rate_limit_can_send(rate_limit)) {
        usleep(1. / (rate_limit->lambda * 1e-6) * 0.01);
    }
}

/* we won't load actual files. just fill the images with random bytes */
void load_image_pairs(uchar *images1, uchar *images2) {
    srand(0);
    for (int i = 0; i < N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION; i++) {
        images1[i] = rand() % 256;
        images2[i] = rand() % 256;
    }
}

__device__ __host__ bool is_in_image_bounds(int i, int j) {
    return (i >= 0) && (i < IMG_DIMENSION) && (j >= 0) && (j < IMG_DIMENSION);
}

__device__ __host__ uchar local_binary_pattern(uchar *image, int i, int j) {
    uchar center = image[i * IMG_DIMENSION + j];
    uchar pattern = 0;
    if (is_in_image_bounds(i - 1, j - 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j - 1)] >= center) << 7;
    if (is_in_image_bounds(i - 1, j    )) pattern |= (image[(i - 1) * IMG_DIMENSION + (j    )] >= center) << 6;
    if (is_in_image_bounds(i - 1, j + 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j + 1)] >= center) << 5;
    if (is_in_image_bounds(i    , j + 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j + 1)] >= center) << 4;
    if (is_in_image_bounds(i + 1, j + 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j + 1)] >= center) << 3;
    if (is_in_image_bounds(i + 1, j    )) pattern |= (image[(i + 1) * IMG_DIMENSION + (j    )] >= center) << 2;
    if (is_in_image_bounds(i + 1, j - 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j - 1)] >= center) << 1;
    if (is_in_image_bounds(i    , j - 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j - 1)] >= center) << 0;
    return pattern;
}

void image_to_histogram(uchar *image, int *histogram) {
    memset(histogram, 0, sizeof(int) * 256);
    for (int i = 0; i < IMG_DIMENSION; i++) {
        for (int j = 0; j < IMG_DIMENSION; j++) {
            uchar pattern = local_binary_pattern(image, i, j);
            histogram[pattern]++;
        }
    }
}

double histogram_distance(int *h1, int *h2) {
    /* we'll use the chi-square distance */
    float distance = 0;
    for (int i = 0; i < 256; i++) {
        if (h1[i] + h2[i] != 0) {
            distance += ((double)SQR(h1[i] - h2[i])) / (h1[i] + h2[i]);
        }
    }
    return distance;
}

__global__ void gpu_image_to_histogram(uchar *image, int *histogram) {
    uchar pattern = local_binary_pattern(image, threadIdx.x / IMG_DIMENSION, threadIdx.x % IMG_DIMENSION);
    atomicAdd(&histogram[pattern], 1);
}

__global__ void gpu_histogram_distance(int *h1, int *h2, double *distance) {
    int length = 256;
    int tid = threadIdx.x;
    distance[tid] = 0;
    if (h1[tid] + h2[tid] != 0) {
        distance[tid] = ((double)SQR(h1[tid] - h2[tid])) / (h1[tid] + h2[tid]);
    }
    __syncthreads();

    while (length > 1) {
        if (threadIdx.x < length / 2) {
            distance[tid] = distance[tid] + distance[tid + length / 2];
        }
        length /= 2;
        __syncthreads();
    }
}

__global__ void process_queues (volatile threads_queue *dev_gpu_cpu_queues, volatile threads_queue *dev_cpu_gpu_queues,
                                const int max_simult_blocks,
                                volatile uchar *gpu_image1,
                                volatile uchar *gpu_image2) {
    __shared__ int          req_id;
    __shared__ bool         was_deq;
    __shared__ unsigned int req_count;
    __shared__ uchar s_image1[IMG_DIMENSION * IMG_DIMENSION], s_image2[IMG_DIMENSION * IMG_DIMENSION];
    __shared__ int s_hist1[256], s_hist2[256];
    __shared__ float s_distance[256];
    __shared__ uchar pattern1[IMG_DIMENSION * IMG_DIMENSION], pattern2[IMG_DIMENSION * IMG_DIMENSION];
    int nrequests = NREQUESTS / max_simult_blocks + !(blockIdx.x >= NREQUESTS % max_simult_blocks);
    int copy_iter = (IMG_DIMENSION * IMG_DIMENSION) / blockDim.x;
    volatile uchar *dcpu_gpu_read_idx  = &dev_cpu_gpu_queues[blockIdx.x].read_index;
    volatile uchar *dcpu_gpu_write_idx = &dev_cpu_gpu_queues[blockIdx.x].write_index;
    volatile uchar *dgpu_cpu_read_idx  = &dev_gpu_cpu_queues[blockIdx.x].read_index;
    volatile uchar *dgpu_cpu_write_idx = &dev_gpu_cpu_queues[blockIdx.x].write_index;

    if (threadIdx.x == 0) {
        was_deq = false; req_count = 0;
    }
    __threadfence_block();

    while (req_count < nrequests) {
        if (threadIdx.x == 0) {
            volatile uchar cpu_gpu_read_idx = *dcpu_gpu_read_idx;
            volatile uchar cpu_gpu_write_idx = *dcpu_gpu_write_idx;
            __threadfence_system();
            // Dequeue request
            if (!was_deq && cpu_gpu_read_idx != cpu_gpu_write_idx) {
                req_id = dev_cpu_gpu_queues[blockIdx.x].queue_array[cpu_gpu_read_idx].req_id;
                *dcpu_gpu_read_idx = (cpu_gpu_read_idx + 1) % QUEUENODES;
                __threadfence_system();
                was_deq = true;
                __threadfence_block();
            }
        }
        __syncthreads();
        // Init data structures for request processing
        if (was_deq) {
            if (threadIdx.x < 256) {
                s_hist1[threadIdx.x] = 0;
                s_hist2[threadIdx.x] = 0;
                s_distance[threadIdx.x] = 0;
            }
            for (int i = 0; i < copy_iter; i++) {
                s_image1[i * blockDim.x + threadIdx.x] = gpu_image1[req_id * IMG_DIMENSION * IMG_DIMENSION + i * blockDim.x + threadIdx.x];
                s_image2[i * blockDim.x + threadIdx.x] = gpu_image2[req_id * IMG_DIMENSION * IMG_DIMENSION + i * blockDim.x + threadIdx.x];
                pattern1[i * blockDim.x + threadIdx.x] = 0;
                pattern2[i * blockDim.x + threadIdx.x] = 0;
            }
        }
        __threadfence_block();

        // Calculate image patterns
        if (was_deq) {
            for (int j = 0; j < copy_iter; j++) {
            	pattern1[j * blockDim.x + threadIdx.x] = local_binary_pattern((uchar *)s_image1, (j * blockDim.x + threadIdx.x) / IMG_DIMENSION, (j * blockDim.x + threadIdx.x) % IMG_DIMENSION);
            }

            for (int j = 0; j < copy_iter; j++) {
            	pattern2[j * blockDim.x + threadIdx.x] = local_binary_pattern((uchar *)s_image2, (j * blockDim.x + threadIdx.x) / IMG_DIMENSION, (j * blockDim.x + threadIdx.x) % IMG_DIMENSION);
            }
        }
        __threadfence_block();

        // Calculate histograms
        if (was_deq) {
        	for (int j = 0; j < copy_iter; j++) {
        		if (pattern1[j * blockDim.x + threadIdx.x] != 0) {
        			atomicAdd((int *)&(s_hist1[pattern1[j * blockDim.x + threadIdx.x]]), 1);
        		}
        		if (pattern2[j * blockDim.x + threadIdx.x] != 0) {
        			atomicAdd((int *)&(s_hist2[pattern2[j * blockDim.x + threadIdx.x]]), 1);
        		}
        		__threadfence_block();
            }
        }
        __threadfence_block();

        if (was_deq) {
            if (threadIdx.x < 256) {
                if (s_hist1[threadIdx.x] + s_hist1[threadIdx.x] != 0) {
                    s_distance[threadIdx.x] = ((float)SQR(s_hist1[threadIdx.x] - s_hist2[threadIdx.x])) / (s_hist1[threadIdx.x] + s_hist2[threadIdx.x]);
                    __threadfence();
                }
                // Comment IF statement above and uncomment s_distance[threadIdx.x] = 1 below
                // It will result average distance 256, I used it for histogram distance validation
                //s_distance[threadIdx.x] = 1;
            }
        }
        __syncthreads();

        if (was_deq) {
            if (threadIdx.x < 256) {
                int length = 256;
                __threadfence_block();
                while (length > 1) {
                    if (threadIdx.x < length / 2) {
                        s_distance[threadIdx.x] = s_distance[threadIdx.x] + s_distance[threadIdx.x + length / 2];
                        __threadfence();
                    }
                    length /= 2;
                    __syncthreads();
                }
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            volatile uchar gpu_cpu_read_idx = *dgpu_cpu_read_idx;
            volatile uchar gpu_cpu_write_idx = *dgpu_cpu_write_idx;
            if (was_deq && (gpu_cpu_read_idx != (gpu_cpu_write_idx + 1) % QUEUENODES)) {
                // Enqueue
                dev_gpu_cpu_queues[blockIdx.x].queue_array[gpu_cpu_write_idx].result = s_distance[0];
                dev_gpu_cpu_queues[blockIdx.x].queue_array[gpu_cpu_write_idx].req_id = req_id;
                __threadfence_system();
                *dgpu_cpu_write_idx = (gpu_cpu_write_idx + 1) % QUEUENODES;
                __threadfence_system();
                req_count++;
                was_deq = false;
            }
        }
        __syncthreads();
    }
}

void print_usage_and_die(char *progname) {
    printf("usage:\n");
    printf("%s streams <load (requests/sec)>\n", progname);
    printf("OR\n");
    printf("%s queue <#threads> <load (requests/sec)>\n", progname);
    exit(1);
}


enum {PROGRAM_MODE_STREAMS = 0, PROGRAM_MODE_QUEUE};
int main(int argc, char *argv[]) {

    int mode = -1;
    int threads_queue_mode = -1; /* valid only when mode = queue */
    double load = 0;
    if (argc < 3) print_usage_and_die(argv[0]);

    if        (!strcmp(argv[1], "streams")) {
        if (argc != 3) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_STREAMS;
        load = atof(argv[2]);
    } else if (!strcmp(argv[1], "queue")) {
        if (argc != 4) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_QUEUE;
        threads_queue_mode = atoi(argv[2]);
        load = atof(argv[3]);
    } else {
        print_usage_and_die(argv[0]);
    }

    uchar *images1; /* we concatenate all images in one huge array */
    uchar *images2;
    CUDA_CHECK( hipHostAlloc(&images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );
    CUDA_CHECK( hipHostAlloc(&images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );

    load_image_pairs(images1, images2);
    double t_start, t_finish;
    double total_distance;
#if 1
    /* using CPU */
    printf("\n=== CPU ===\n");
    int histogram1[256];
    int histogram2[256];
    t_start  = get_time_msec();
    for (int i = 0; i < NREQUESTS; i++) {
        int img_idx = i % N_IMG_PAIRS;
        image_to_histogram(&images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram1);
        image_to_histogram(&images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram2);
        total_distance += histogram_distance(histogram1, histogram2);
    }
    t_finish = get_time_msec();
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

    /* using GPU task-serial.. just to verify the GPU code makes sense */
    printf("\n=== GPU Task Serial ===\n");
    do {
        uchar *gpu_image1, *gpu_image2; // TODO: allocate with cudaMalloc
        int *gpu_hist1, *gpu_hist2; // TODO: allocate with cudaMalloc
        double *gpu_hist_distance; //TODO: allocate with cudaMalloc
        double cpu_hist_distance;
        hipMalloc(&gpu_image1, IMG_DIMENSION * IMG_DIMENSION);
        hipMalloc(&gpu_image2, IMG_DIMENSION * IMG_DIMENSION);
        hipMalloc(&gpu_hist1, 256 * sizeof(int));
        hipMalloc(&gpu_hist2, 256 * sizeof(int));
        hipMalloc(&gpu_hist_distance, 256 * sizeof(double));

        total_distance = 0;
        t_start = get_time_msec();
        for (int i = 0; i < NREQUESTS; i++) {
            int img_idx = i % N_IMG_PAIRS;
            hipMemcpy(gpu_image1, &images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice);
            hipMemcpy(gpu_image2, &images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice);
            hipMemset(gpu_hist1, 0, 256 * sizeof(int));
            hipMemset(gpu_hist2, 0, 256 * sizeof(int));
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image1, gpu_hist1);
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image2, gpu_hist2);
            gpu_histogram_distance<<<1, 256>>>(gpu_hist1, gpu_hist2, gpu_hist_distance);
            hipMemcpy(&cpu_hist_distance, gpu_hist_distance, sizeof(double), hipMemcpyDeviceToHost);
            total_distance += cpu_hist_distance;
            //printf("CPU: Req #%d, Result = %lf\n", i, cpu_hist_distance);
        }
        CUDA_CHECK(hipDeviceSynchronize());
        t_finish = get_time_msec();
        printf("average distance between images %f\n", total_distance / NREQUESTS);
        printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);
        CUDA_CHECK( hipFree(gpu_image1) );
        CUDA_CHECK( hipFree(gpu_image2) );
        CUDA_CHECK( hipFree(gpu_hist1) );
        CUDA_CHECK( hipFree(gpu_hist2) );
        CUDA_CHECK( hipFree(gpu_hist_distance) );
    } while (0);
#endif
    /* now for the client-server part */
    printf("\n=== Client-Server ===\n");
    total_distance = 0;
    double *req_t_start = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_start, 0, NREQUESTS * sizeof(double));

    double *req_t_end = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_end, 0, NREQUESTS * sizeof(double));

    struct rate_limit_t rate_limit;
    rate_limit_init(&rate_limit, load, 0);

    /* TODO allocate / initialize memory, streams, etc... */
    uchar *gpu_image1, *gpu_image2; // TODO: allocate with cudaMalloc
    int *gpu_hist1 = NULL, *gpu_hist2 = NULL; // TODO: allocate with cudaMalloc
    double *gpu_hist_distance = NULL; //TODO: allocate with cudaMalloc
    double *cpu_hist_distance;

    streamNode streams_array[NSTREAMS] = {0};
    int free_streams = NSTREAMS;

    double ti = get_time_msec();
    if (mode == PROGRAM_MODE_STREAMS) {
        // Allocate CUDA memory for STREAMS
        CUDA_CHECK(hipMalloc(&gpu_image1, IMG_DIMENSION * IMG_DIMENSION * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_image2, IMG_DIMENSION * IMG_DIMENSION * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_hist1, 256 * sizeof(int) * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_hist2, 256 * sizeof(int) * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_hist_distance, 256 * sizeof(double) * NSTREAMS));
        CUDA_CHECK(hipHostAlloc(&cpu_hist_distance, sizeof(double), 0));

        // Init array of stream nodes
        for (int j = 0; j < NSTREAMS; j++) {
            streams_array[j].stream_id = j;
            streams_array[j].req_in_processing = -1;
            CUDA_CHECK( hipStreamCreate(&streams_array[j].Stream));
        }

        for (int i = 0; i < NREQUESTS; i++) {

            /* TODO query (don't block) streams for any completed requests.
               update req_t_end of completed requests
               update total_distance */

            for (int j = 0; j < NSTREAMS; j++) {
                if ( streams_array[j].req_in_processing != -1) {
                    if ( hipStreamQuery(streams_array[j].Stream) == hipSuccess) {
                        req_t_end[streams_array[j].req_in_processing] = get_time_msec();
                        CUDA_CHECK( hipMemcpyAsync(cpu_hist_distance, &(gpu_hist_distance[streams_array[j].stream_id * 256]), sizeof(double), hipMemcpyDeviceToHost, streams_array[j].Stream));
                        total_distance += *cpu_hist_distance;
                        streams_array[j].req_in_processing = -1;
                        free_streams++;
                    }
                }
            }

            rate_limit_wait(&rate_limit);
            req_t_start[i] = get_time_msec();
            int img_idx = i % N_IMG_PAIRS;

            /* TODO place memcpy's and kernels in a stream */
            if (free_streams > 0) {

                // Find first free stream
                streamNode *busy_streams;
                for (int j = 0; j < NSTREAMS; j++) {
                    if (streams_array[j].req_in_processing == -1) {
                        busy_streams = &streams_array[j];
                        break;
                    }
                }

                busy_streams->req_in_processing = i;
                free_streams--;

                // Enqueue data copy and kernel execution for selected stream
                CUDA_CHECK(hipMemcpyAsync(&(gpu_image1[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice, busy_streams->Stream));
                CUDA_CHECK(hipMemcpyAsync(&(gpu_image2[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice, busy_streams->Stream));
                CUDA_CHECK(hipMemsetAsync(&(gpu_hist1[busy_streams->stream_id * 256]), 0, 256 * sizeof(int), busy_streams->Stream));
                CUDA_CHECK(hipMemsetAsync(&(gpu_hist2[busy_streams->stream_id * 256]), 0, 256 * sizeof(int), busy_streams->Stream));

                gpu_image_to_histogram<<<1, 1024, 0, busy_streams->Stream>>>(&(gpu_image1[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &(gpu_hist1[busy_streams->stream_id * 256]));
                gpu_image_to_histogram<<<1, 1024, 0, busy_streams->Stream>>>(&(gpu_image2[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &(gpu_hist2[busy_streams->stream_id * 256]));
                gpu_histogram_distance<<<1, 256, 0, busy_streams->Stream>>>(&(gpu_hist1[busy_streams->stream_id * 256]), &(gpu_hist2[busy_streams->stream_id * 256]), &(gpu_hist_distance[busy_streams->stream_id * 256]));
            }
        }
        /* TODO now make sure to wait for all streams to finish */
        for (int j = 0; j < NSTREAMS; j++) {
            if (streams_array[j].req_in_processing != -1) {
                CUDA_CHECK( hipStreamSynchronize(streams_array[j].Stream) );
                req_t_end[streams_array[j].req_in_processing] = get_time_msec();
            }
        }

        for (int j = 0; j < NSTREAMS; j++) {
            CUDA_CHECK( hipStreamDestroy(streams_array[j].Stream));
        }

        CUDA_CHECK( hipFree(gpu_hist_distance) );
        CUDA_CHECK( hipHostFree(cpu_hist_distance) );

    } else if (mode == PROGRAM_MODE_QUEUE) {
        // Check for CUDA device and calculate amount of CPU<->GPU queues accordingly to it's capabilities
        int deviceCount = 0, cuda_device = 0;
        CUDA_CHECK( hipGetDeviceCount(&deviceCount) );
        if (deviceCount > 0) {
            printf("CUDA Device(s) found, will use first available device: ");
        } else {
            printf("No CUDA Device found, terminating the program!\n");
            assert(0);
        }
        hipSetDevice(cuda_device);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, cuda_device);
        printf("%s\n", deviceProp.name);
        if (	threads_queue_mode > IMG_DIMENSION * IMG_DIMENSION  ||
                threads_queue_mode <= 0    ||
                deviceProp.maxThreadsPerBlock < IMG_DIMENSION * IMG_DIMENSION
            )
        {
            printf("Wrong amount of threads requested for 32x32 images or your device incapable to run 1024 threads in one block,\nPlease enter #threads = 1024 or less.\n");
            assert (0);
        }

        if ( deviceProp.canMapHostMemory == 0){
            printf("Your CUDA Device doesn't support cudaDeviceMapHost, terminating the program!\n");
            assert(0);
        }

        unsigned int max_simult_blocks = deviceProp.multiProcessorCount * (deviceProp.maxThreadsPerMultiProcessor / threads_queue_mode);
        printf("This device is capable to run %d thread blocks (TB) simultaneously.\n", max_simult_blocks);
        if ( (deviceProp.regsPerBlock / (max_simult_blocks * threads_queue_mode)) < MAXREGCOUNT) {
            max_simult_blocks = deviceProp.regsPerBlock / ( threads_queue_mode * MAXREGCOUNT );
            printf("Amount of running TBs (queue pairs) was reduced to %d due to GPU Registers limitation per TB\n", max_simult_blocks);
        }
        if ( (deviceProp.sharedMemPerBlock / (2 * (IMG_DIMENSION * IMG_DIMENSION) + 2 * sizeof (int) * 256 + sizeof (double) * 256)) < 1) {
            printf("No enough Shared memory per block, terminating the program!\n");
            assert(0);
        }

        // Create CPU<->GPU queues
        volatile threads_queue *cpu_gpu_queues;
        volatile threads_queue *gpu_cpu_queues;
        CUDA_CHECK( hipHostAlloc(&gpu_cpu_queues, sizeof (threads_queue) * max_simult_blocks, hipHostMallocMapped) );
        CUDA_CHECK( hipHostAlloc(&cpu_gpu_queues, sizeof (threads_queue) * max_simult_blocks, hipHostMallocMapped) );

        for (int i = 0; i < max_simult_blocks; i++) {
            cpu_gpu_queues[i].read_index = 0;
            cpu_gpu_queues[i].write_index = 0;
            gpu_cpu_queues[i].read_index = 0;
            gpu_cpu_queues[i].write_index = 0;
        }

        CUDA_CHECK( hipMalloc(&gpu_image1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION) );
        CUDA_CHECK( hipMalloc(&gpu_image2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION) );

        CUDA_CHECK( hipMemcpy(gpu_image1, images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice) );
        CUDA_CHECK( hipMemcpy(gpu_image2, images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice) );

        volatile threads_queue *dev_cpu_gpu_queues, *dev_gpu_cpu_queues;
        CUDA_CHECK( hipHostGetDevicePointer ((void **)&dev_gpu_cpu_queues, (void *)gpu_cpu_queues, 0) );
        CUDA_CHECK( hipHostGetDevicePointer ((void **)&dev_cpu_gpu_queues, (void *)cpu_gpu_queues, 0) );
        CUDA_CHECK( hipDeviceSynchronize() );

        //Start CUDA kernel
        process_queues<<<max_simult_blocks, threads_queue_mode>>>(  dev_gpu_cpu_queues,
                                                                    dev_cpu_gpu_queues,
                                                                    max_simult_blocks,
                                                                    gpu_image1,
                                                                    gpu_image2);

        for (unsigned int i = 0, k = 0; k < NREQUESTS ;) {

            /* TODO check producer consumer queue for any responses.
               don't block. if no responses are there we'll check again in the next iteration
               update req_t_end of completed requests
               update total_distance */
            for (int j = 0; j < max_simult_blocks; j++) {
                // Get current GPU->CPU queue indexes
                volatile uchar *pgpu_cpu_read_index = &gpu_cpu_queues[j].read_index;
                volatile uchar *pgpu_cpu_write_index = &gpu_cpu_queues[j].write_index;
                volatile uchar read_idx  = *pgpu_cpu_read_index;
                volatile uchar write_idx = *pgpu_cpu_write_index;

                volatile int req_id = gpu_cpu_queues[j].queue_array[read_idx].req_id;
                volatile double result = gpu_cpu_queues[j].queue_array[read_idx].result;

                // Dequeue completed
                if (read_idx != write_idx) {
                    total_distance += result;
                    req_t_end[req_id] = get_time_msec();
                    *pgpu_cpu_read_index = (read_idx + 1) % QUEUENODES;
                    __sync_synchronize();
                    // Advance completed requests counter
                    k++;
                }
            }


            if (i < NREQUESTS) {
                rate_limit_wait(&rate_limit);
                int queue_idx = i % max_simult_blocks;
                req_t_start[i] = get_time_msec();
                volatile uchar read_idx = cpu_gpu_queues[queue_idx].read_index;
                volatile uchar write_idx = cpu_gpu_queues[queue_idx].write_index;
                volatile uchar *pcpu_gpu_write_index = &cpu_gpu_queues[queue_idx].write_index;
                if (read_idx != (write_idx + 1) % QUEUENODES) {
                    // Enqueue
                    cpu_gpu_queues[queue_idx].queue_array[write_idx].req_id = i;
                    *pcpu_gpu_write_index = (write_idx + 1) % QUEUENODES;
                    __sync_synchronize();
                    //printf("CPU: CPU-GPU write index #%d was increased by thread %d\n", write_idx, queue_idx);
                    // Advance request id
                    i++;
                }
            }
        }
        /* TODO wait until you have responses for all requests */
        CUDA_CHECK( hipDeviceSynchronize() );

        // Release memory allocations specific for threads flow
        CUDA_CHECK( hipHostFree((void *)gpu_cpu_queues) );
        CUDA_CHECK( hipHostFree((void *)cpu_gpu_queues) );
    } else {
        assert(0);
    }
    double tf = get_time_msec();

    CUDA_CHECK( hipFree(gpu_image1) );
    CUDA_CHECK( hipFree(gpu_image2) );
    if (gpu_hist1 != NULL) CUDA_CHECK( hipFree(gpu_hist1) );
    if (gpu_hist2 != NULL) CUDA_CHECK( hipFree(gpu_hist2) );
    if (gpu_hist_distance != NULL) CUDA_CHECK( hipFree(gpu_hist_distance) );

    double avg_latency = 0;
    for (int j = 0; j < NREQUESTS; j++) {
        avg_latency += (req_t_end[j] - req_t_start[j]);
    }
    avg_latency /= NREQUESTS;

    printf("mode = %s\n", mode == PROGRAM_MODE_STREAMS ? "streams" : "queue");
    printf("load = %lf (req/sec)\n", load);
    if (mode == PROGRAM_MODE_QUEUE) printf("threads = %d\n", threads_queue_mode);
    printf("average distance between images %lf\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (tf - ti) * 1e+3);
    printf("average latency = %lf (msec)\n", avg_latency);

    return 0;
}
