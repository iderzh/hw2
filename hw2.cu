/* compile with: nvcc -O3 -maxrregcount=32 hw2.cu -o hw2 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <assert.h>
#include <string.h>

#define IMG_DIMENSION 32
#define N_IMG_PAIRS 10000
#define NREQUESTS 100000
#define NSTREAMS 64

typedef unsigned char uchar;
#define OUT

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}

struct stream_node {
	hipStream_t Stream;
	int stream_id;
	int req_in_processing;
};
typedef stream_node streamNode;

/* we'll use these to rate limit the request load */
struct rate_limit_t {
    double last_checked;
    double lambda;
    unsigned seed;
};

void rate_limit_init(struct rate_limit_t *rate_limit, double lambda, int seed) {
    rate_limit->lambda = lambda;
    rate_limit->seed = (seed == -1) ? 0 : seed;
    rate_limit->last_checked = 0;
}

int rate_limit_can_send(struct rate_limit_t *rate_limit) {
    if (rate_limit->lambda == 0) return 1;
    double now = get_time_msec() * 1e-3;
    double dt = now - rate_limit->last_checked;
    double p = dt * rate_limit->lambda;
    rate_limit->last_checked = now;
    if (p > 1) p = 1;
    double r = (double)rand_r(&rate_limit->seed) / RAND_MAX;
    return (p > r);
}

void rate_limit_wait(struct rate_limit_t *rate_limit) {
    while (!rate_limit_can_send(rate_limit)) {
        usleep(1. / (rate_limit->lambda * 1e-6) * 0.01);
    }
}

/* we won't load actual files. just fill the images with random bytes */
void load_image_pairs(uchar *images1, uchar *images2) {
    srand(0);
    for (int i = 0; i < N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION; i++) {
        images1[i] = rand() % 256;
        images2[i] = rand() % 256;
    }
}

__device__ __host__ bool is_in_image_bounds(int i, int j) {
    return (i >= 0) && (i < IMG_DIMENSION) && (j >= 0) && (j < IMG_DIMENSION);
}

__device__ __host__ uchar local_binary_pattern(uchar *image, int i, int j) {
    uchar center = image[i * IMG_DIMENSION + j];
    uchar pattern = 0;
    if (is_in_image_bounds(i - 1, j - 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j - 1)] >= center) << 7;
    if (is_in_image_bounds(i - 1, j    )) pattern |= (image[(i - 1) * IMG_DIMENSION + (j    )] >= center) << 6;
    if (is_in_image_bounds(i - 1, j + 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j + 1)] >= center) << 5;
    if (is_in_image_bounds(i    , j + 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j + 1)] >= center) << 4;
    if (is_in_image_bounds(i + 1, j + 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j + 1)] >= center) << 3;
    if (is_in_image_bounds(i + 1, j    )) pattern |= (image[(i + 1) * IMG_DIMENSION + (j    )] >= center) << 2;
    if (is_in_image_bounds(i + 1, j - 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j - 1)] >= center) << 1;
    if (is_in_image_bounds(i    , j - 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j - 1)] >= center) << 0;
    return pattern;
}

void image_to_histogram(uchar *image, int *histogram) {
    memset(histogram, 0, sizeof(int) * 256);
    for (int i = 0; i < IMG_DIMENSION; i++) {
        for (int j = 0; j < IMG_DIMENSION; j++) {
            uchar pattern = local_binary_pattern(image, i, j);
            histogram[pattern]++;
        }
    }
}

double histogram_distance(int *h1, int *h2) {
    /* we'll use the chi-square distance */
    float distance = 0;
    for (int i = 0; i < 256; i++) {
        if (h1[i] + h2[i] != 0) {
            distance += ((double)SQR(h1[i] - h2[i])) / (h1[i] + h2[i]);
        }
    }
    return distance;
}

__global__ void gpu_image_to_histogram(uchar *image, int *histogram) {
    uchar pattern = local_binary_pattern(image, threadIdx.x / IMG_DIMENSION, threadIdx.x % IMG_DIMENSION);
    atomicAdd(&histogram[pattern], 1);
}

__global__ void gpu_histogram_distance(int *h1, int *h2, double *distance) {
    int length = 256;
    int tid = threadIdx.x;
    distance[tid] = 0;
    if (h1[tid] + h2[tid] != 0) {
        distance[tid] = ((double)SQR(h1[tid] - h2[tid])) / (h1[tid] + h2[tid]);
    }
    __syncthreads();

    while (length > 1) {
        if (threadIdx.x < length / 2) {
            distance[tid] = distance[tid] + distance[tid + length / 2];
        }
        length /= 2;
        __syncthreads();
    }
}

void print_usage_and_die(char *progname) {
    printf("usage:\n");
    printf("%s streams <load (requests/sec)>\n", progname);
    printf("OR\n");
    printf("%s queue <#threads> <load (requests/sec)>\n", progname);
    exit(1);
}


enum {PROGRAM_MODE_STREAMS = 0, PROGRAM_MODE_QUEUE};
int main(int argc, char *argv[]) {

    int mode = -1;
    int threads_queue_mode = -1; /* valid only when mode = queue */
    double load = 0;
    if (argc < 3) print_usage_and_die(argv[0]);

    if        (!strcmp(argv[1], "streams")) {
        if (argc != 3) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_STREAMS;
        load = atof(argv[2]);
    } else if (!strcmp(argv[1], "queue")) {
        if (argc != 4) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_QUEUE;
        threads_queue_mode = atoi(argv[2]);
        load = atof(argv[3]);
    } else {
        print_usage_and_die(argv[0]);
    }

    uchar *images1; /* we concatenate all images in one huge array */
    uchar *images2;
    CUDA_CHECK( hipHostAlloc(&images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );
    CUDA_CHECK( hipHostAlloc(&images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );

    load_image_pairs(images1, images2);
    double t_start, t_finish;
    double total_distance;
#if 1
    /* using CPU */
    printf("\n=== CPU ===\n");
    int histogram1[256];
    int histogram2[256];
    t_start  = get_time_msec();
    for (int i = 0; i < NREQUESTS; i++) {
        int img_idx = i % N_IMG_PAIRS;
        image_to_histogram(&images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram1);
        image_to_histogram(&images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram2);
        total_distance += histogram_distance(histogram1, histogram2);
    }
    t_finish = get_time_msec();
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

    /* using GPU task-serial.. just to verify the GPU code makes sense */
    printf("\n=== GPU Task Serial ===\n");
    do {
        uchar *gpu_image1, *gpu_image2; // TODO: allocate with cudaMalloc
        int *gpu_hist1, *gpu_hist2; // TODO: allocate with cudaMalloc
        double *gpu_hist_distance; //TODO: allocate with cudaMalloc
        double cpu_hist_distance;
        hipMalloc(&gpu_image1, IMG_DIMENSION * IMG_DIMENSION);
        hipMalloc(&gpu_image2, IMG_DIMENSION * IMG_DIMENSION);
        hipMalloc(&gpu_hist1, 256 * sizeof(int));
        hipMalloc(&gpu_hist2, 256 * sizeof(int));
        hipMalloc(&gpu_hist_distance, 256 * sizeof(double));

        total_distance = 0;
        t_start = get_time_msec();
        for (int i = 0; i < NREQUESTS; i++) {
            int img_idx = i % N_IMG_PAIRS;
            hipMemcpy(gpu_image1, &images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice);
            hipMemcpy(gpu_image2, &images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice);
            hipMemset(gpu_hist1, 0, 256 * sizeof(int));
            hipMemset(gpu_hist2, 0, 256 * sizeof(int));
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image1, gpu_hist1);
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image2, gpu_hist2);
            gpu_histogram_distance<<<1, 256>>>(gpu_hist1, gpu_hist2, gpu_hist_distance);
            hipMemcpy(&cpu_hist_distance, gpu_hist_distance, sizeof(double), hipMemcpyDeviceToHost);
            total_distance += cpu_hist_distance;
        }
        CUDA_CHECK(hipDeviceSynchronize());
        t_finish = get_time_msec();
        printf("average distance between images %f\n", total_distance / NREQUESTS);
        printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);
        CUDA_CHECK( hipFree(gpu_image1) );
        CUDA_CHECK( hipFree(gpu_image2) );
        CUDA_CHECK( hipFree(gpu_hist1) );
        CUDA_CHECK( hipFree(gpu_hist2) );
        CUDA_CHECK( hipFree(gpu_hist_distance) );
    } while (0);
#endif
    /* now for the client-server part */
    printf("\n=== Client-Server ===\n");
    total_distance = 0;
    double *req_t_start = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_start, 0, NREQUESTS * sizeof(double));

    double *req_t_end = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_end, 0, NREQUESTS * sizeof(double));

    struct rate_limit_t rate_limit;
    rate_limit_init(&rate_limit, load, 0);

    /* TODO allocate / initialize memory, streams, etc... */
    uchar *gpu_image1, *gpu_image2; // TODO: allocate with cudaMalloc
    int *gpu_hist1, *gpu_hist2; // TODO: allocate with cudaMalloc
    double *gpu_hist_distance; //TODO: allocate with cudaMalloc
    double *cpu_hist_distance;

    streamNode streams_array[NSTREAMS] = {0};
    int free_streams = NSTREAMS;

    CUDA_CHECK(hipMalloc(&gpu_image1, IMG_DIMENSION * IMG_DIMENSION * NSTREAMS));
    CUDA_CHECK(hipMalloc(&gpu_image2, IMG_DIMENSION * IMG_DIMENSION * NSTREAMS));
    CUDA_CHECK(hipMalloc(&gpu_hist1, 256 * sizeof(int) * NSTREAMS));
    CUDA_CHECK(hipMalloc(&gpu_hist2, 256 * sizeof(int) * NSTREAMS));
    CUDA_CHECK(hipMalloc(&gpu_hist_distance, 256 * sizeof(double) * NSTREAMS));
    CUDA_CHECK(hipHostAlloc(&cpu_hist_distance, sizeof(double), 0));

    double ti = get_time_msec();
    if (mode == PROGRAM_MODE_STREAMS) {

        // Init array of stream nodes
        for (int j = 0; j < NSTREAMS; j++) {
        	streams_array[j].stream_id = j;
        	streams_array[j].req_in_processing = -1;
        	CUDA_CHECK( hipStreamCreate(&streams_array[j].Stream));
        }

        for (int i = 0; i < NREQUESTS; i++) {

            /* TODO query (don't block) streams for any completed requests.
               update req_t_end of completed requests
               update total_distance */

        	for (int j = 0; j < NSTREAMS; j++) {
        		if ( streams_array[j].req_in_processing != -1) {
        			if ( hipStreamQuery(streams_array[j].Stream) == hipSuccess) {
        				req_t_end[streams_array[j].req_in_processing] = get_time_msec();
        				CUDA_CHECK( hipMemcpyAsync(cpu_hist_distance, &(gpu_hist_distance[streams_array[j].stream_id * 256]), sizeof(double), hipMemcpyDeviceToHost, streams_array[j].Stream));
        				total_distance += *cpu_hist_distance;
        				streams_array[j].req_in_processing = -1;
        				free_streams++;
        			}
        		}
        	}

            rate_limit_wait(&rate_limit);
            req_t_start[i] = get_time_msec();
            int img_idx = i % N_IMG_PAIRS;

            /* TODO place memcpy's and kernels in a stream */
            if (free_streams > 0) {

            	// Find first free stream
            	streamNode *busy_streams;
            	for (int j = 0; j < NSTREAMS; j++) {
            		if (streams_array[j].req_in_processing == -1) {
            			busy_streams = &streams_array[j];
            			break;
            		}
            	}

            	busy_streams->req_in_processing = i;
            	free_streams--;

                // Enqueue data copy and kernel execution for selected stream
                CUDA_CHECK(hipMemcpyAsync(&(gpu_image1[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice, busy_streams->Stream));
                CUDA_CHECK(hipMemcpyAsync(&(gpu_image2[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice, busy_streams->Stream));
                CUDA_CHECK(hipMemsetAsync(&(gpu_hist1[busy_streams->stream_id * 256]), 0, 256 * sizeof(int), busy_streams->Stream));
                CUDA_CHECK(hipMemsetAsync(&(gpu_hist2[busy_streams->stream_id * 256]), 0, 256 * sizeof(int), busy_streams->Stream));

                gpu_image_to_histogram<<<1, 1024, 0, busy_streams->Stream>>>(&(gpu_image1[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &(gpu_hist1[busy_streams->stream_id * 256]));
                gpu_image_to_histogram<<<1, 1024, 0, busy_streams->Stream>>>(&(gpu_image2[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &(gpu_hist2[busy_streams->stream_id * 256]));
                gpu_histogram_distance<<<1, 256, 0, busy_streams->Stream>>>(&(gpu_hist1[busy_streams->stream_id * 256]), &(gpu_hist2[busy_streams->stream_id * 256]), &(gpu_hist_distance[busy_streams->stream_id * 256]));
            }
        }
        /* TODO now make sure to wait for all streams to finish */
    	for (int j = 0; j < NSTREAMS; j++) {
    		if (streams_array[j].req_in_processing != -1) {
    			CUDA_CHECK( hipStreamSynchronize(streams_array[j].Stream) );
    			req_t_end[streams_array[j].req_in_processing] = get_time_msec();
    		}
    	}

        for (int j = 0; j < NSTREAMS; j++) {
        	CUDA_CHECK( hipStreamDestroy(streams_array[j].Stream));
        }

    } else if (mode == PROGRAM_MODE_QUEUE) {
        for (int i = 0; i < NREQUESTS; i++) {

            /* TODO check producer consumer queue for any responses.
               don't block. if no responses are there we'll check again in the next iteration
               update req_t_end of completed requests
               update total_distance */

            rate_limit_wait(&rate_limit);
            int img_idx = i % N_IMG_PAIRS;
            req_t_start[i] = get_time_msec();

            /* TODO place memcpy's and kernels in a stream */
        }
        /* TODO wait until you have responses for all requests */
    } else {
        assert(0);
    }
    double tf = get_time_msec();

    CUDA_CHECK( hipFree(gpu_image1) );
    CUDA_CHECK( hipFree(gpu_image2) );
    CUDA_CHECK( hipFree(gpu_hist1) );
    CUDA_CHECK( hipFree(gpu_hist2) );
    CUDA_CHECK( hipFree(gpu_hist_distance) );
    CUDA_CHECK( hipHostFree(cpu_hist_distance) );

    double avg_latency = 0;
    for (int j = 0; j < NREQUESTS; j++) {
        avg_latency += (req_t_end[j] - req_t_start[j]);
    }
    avg_latency /= NREQUESTS;

    printf("mode = %s\n", mode == PROGRAM_MODE_STREAMS ? "streams" : "queue");
    printf("load = %lf (req/sec)\n", load);
    if (mode == PROGRAM_MODE_QUEUE) printf("threads = %d\n", threads_queue_mode);
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (tf - ti) * 1e+3);
    printf("average latency = %lf (msec)\n", avg_latency);

    return 0;
}
