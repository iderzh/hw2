#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "device_functions.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <assert.h>
#include <windows.h>
#include <intrin.h>
#else
/* compile with: nvcc -O3 -maxrregcount=32 hw2.cu -o hw2 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <assert.h>
#include <string.h>
#endif

#define IMG_DIMENSION 32
#define N_IMG_PAIRS 10000
#define NREQUESTS 1234
#define NSTREAMS 64
#define MAXREGCOUNT 32
#define QUEUENODES 10

typedef unsigned char uchar;
#define OUT

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define CUDA_CHECK(f) do {                                                                  \
    cudaError_t e = f;                                                                      \
    if (e != cudaSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, cudaGetErrorString(e));    \
        return 1;                                                                           \
    }                                                                                       \
} while (0)
#else
#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)
#endif

#define SQR(a) ((a) * (a))

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#if !defined(WIN32_LEAN_AND_MEAN)
#define WIN32_LEAN_AND_MEAN
#endif
#define __sync_synchronize() _ReadWriteBarrier()
double static inline get_time_msec(void) {
    LARGE_INTEGER t;
    static double oofreq;
    static int checkedForHighResTimer;
    static BOOL hasHighResTimer;

    if (!checkedForHighResTimer)
    {
        hasHighResTimer = QueryPerformanceFrequency(&t);
        oofreq = 1000.0 / (double)t.QuadPart;
        checkedForHighResTimer = 1;
    }

    if (hasHighResTimer)
    {
        QueryPerformanceCounter(&t);
        return (double)t.QuadPart * oofreq;
    }
    else
    {
        return (double)GetTickCount();
    }
}
void usleep(unsigned int usec)
{
    HANDLE timer;
    LARGE_INTEGER ft;

    ft.QuadPart = -(10 * (__int64)usec);

    timer = CreateWaitableTimer(NULL, TRUE, NULL);
    SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
    WaitForSingleObject(timer, INFINITE);
    CloseHandle(timer);
}
int rand_r(unsigned int *pseed) {
    srand(*pseed);
    return rand();
}
#else
double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}
#endif
struct stream_node {
	hipStream_t Stream;
	int stream_id;
	int req_in_processing;
};
typedef stream_node streamNode;

typedef struct _thread_node {
	int req_id;
	float result;
} thread_node;

typedef struct _threads_queue {
	volatile uchar read_index;
    volatile uchar write_index;
    volatile thread_node queue_array[QUEUENODES];
} threads_queue;

/* we'll use these to rate limit the request load */
struct rate_limit_t {
    double last_checked;
    double lambda;
    unsigned seed;
};

void rate_limit_init(struct rate_limit_t *rate_limit, double lambda, int seed) {
    rate_limit->lambda = lambda;
    rate_limit->seed = (seed == -1) ? 0 : seed;
    rate_limit->last_checked = 0;
}

int rate_limit_can_send(struct rate_limit_t *rate_limit) {
    if (rate_limit->lambda == 0) return 1;
    double now = get_time_msec() * 1e-3;
    double dt = now - rate_limit->last_checked;
    double p = dt * rate_limit->lambda;
    rate_limit->last_checked = now;
    if (p > 1) p = 1;
    double r = (double)rand_r(&rate_limit->seed) / RAND_MAX;
    return (p > r);
}

void rate_limit_wait(struct rate_limit_t *rate_limit) {
    while (!rate_limit_can_send(rate_limit)) {
        usleep(1. / (rate_limit->lambda * 1e-6) * 0.01);
    }
}

/* we won't load actual files. just fill the images with random bytes */
void load_image_pairs(uchar *images1, uchar *images2) {
    srand(0);
    for (int i = 0; i < N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION; i++) {
        images1[i] = rand() % 256;
        images2[i] = rand() % 256;
    }
}

__device__ __host__ bool is_in_image_bounds(int i, int j) {
    return (i >= 0) && (i < IMG_DIMENSION) && (j >= 0) && (j < IMG_DIMENSION);
}

__device__ __host__ uchar local_binary_pattern(uchar *image, int i, int j) {
    uchar center = image[i * IMG_DIMENSION + j];
    uchar pattern = 0;
    if (is_in_image_bounds(i - 1, j - 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j - 1)] >= center) << 7;
    if (is_in_image_bounds(i - 1, j    )) pattern |= (image[(i - 1) * IMG_DIMENSION + (j    )] >= center) << 6;
    if (is_in_image_bounds(i - 1, j + 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j + 1)] >= center) << 5;
    if (is_in_image_bounds(i    , j + 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j + 1)] >= center) << 4;
    if (is_in_image_bounds(i + 1, j + 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j + 1)] >= center) << 3;
    if (is_in_image_bounds(i + 1, j    )) pattern |= (image[(i + 1) * IMG_DIMENSION + (j    )] >= center) << 2;
    if (is_in_image_bounds(i + 1, j - 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j - 1)] >= center) << 1;
    if (is_in_image_bounds(i    , j - 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j - 1)] >= center) << 0;
    return pattern;
}

void image_to_histogram(uchar *image, int *histogram) {
    memset(histogram, 0, sizeof(int) * 256);
    for (int i = 0; i < IMG_DIMENSION; i++) {
        for (int j = 0; j < IMG_DIMENSION; j++) {
            uchar pattern = local_binary_pattern(image, i, j);
            histogram[pattern]++;
        }
    }
}

double histogram_distance(int *h1, int *h2) {
    /* we'll use the chi-square distance */
    float distance = 0;
    for (int i = 0; i < 256; i++) {
        if (h1[i] + h2[i] != 0) {
            distance += ((double)SQR(h1[i] - h2[i])) / (h1[i] + h2[i]);
        }
    }
    return distance;
}

__global__ void gpu_image_to_histogram(uchar *image, int *histogram) {
    uchar pattern = local_binary_pattern(image, threadIdx.x / IMG_DIMENSION, threadIdx.x % IMG_DIMENSION);
    atomicAdd(&histogram[pattern], 1);
}

__global__ void gpu_histogram_distance(int *h1, int *h2, double *distance) {
    int length = 256;
    int tid = threadIdx.x;
    distance[tid] = 0;
    if (h1[tid] + h2[tid] != 0) {
        distance[tid] = ((double)SQR(h1[tid] - h2[tid])) / (h1[tid] + h2[tid]);
    }
    __syncthreads();

    while (length > 1) {
        if (threadIdx.x < length / 2) {
            distance[tid] = distance[tid] + distance[tid + length / 2];
        }
        length /= 2;
        __syncthreads();
    }
}

__global__ void process_queues (volatile threads_queue *dev_gpu_cpu_queues, volatile threads_queue *dev_cpu_gpu_queues, const signed int max_simult_blocks) {
	int req_id = -1;
    __shared__ bool         was_deq;
    __shared__ unsigned int req_count;
    volatile uchar *dcpu_gpu_read_idx  = &dev_cpu_gpu_queues[blockIdx.x].read_index;
    volatile uchar *dcpu_gpu_write_idx = &dev_cpu_gpu_queues[blockIdx.x].write_index;
    volatile uchar *dgpu_cpu_read_idx  = &dev_gpu_cpu_queues[blockIdx.x].read_index;
    volatile uchar *dgpu_cpu_write_idx = &dev_gpu_cpu_queues[blockIdx.x].write_index;
    int nrequests = NREQUESTS / max_simult_blocks + !(blockIdx.x >= NREQUESTS % max_simult_blocks);

    if (threadIdx.x == 0) {
        was_deq = false; req_count = 0;
        //printf("GPU: nrequests = %d in TB #%d\n", nrequests, blockIdx.x);
    }
    __threadfence_system();

    while (req_count < nrequests) {
        if (threadIdx.x == 0) {
            volatile uchar cpu_gpu_read_idx = *dcpu_gpu_read_idx;
            volatile uchar cpu_gpu_write_idx = *dcpu_gpu_write_idx;
            __threadfence_system();
            // Dequeue request
            if (!was_deq && cpu_gpu_read_idx != cpu_gpu_write_idx) {
                req_id = dev_cpu_gpu_queues[blockIdx.x].queue_array[cpu_gpu_read_idx].req_id;
                *dcpu_gpu_read_idx = (cpu_gpu_read_idx + 1) % QUEUENODES;
                was_deq = true;
                __threadfence_system();
                //printf("GPUp: Req #%d was dequeued in TB #%d by thread %d, RC = %d\n", req_id, blockIdx.x, threadIdx.x, req_count);
            };
        }
        //__threadfence();
        __syncthreads();


        if (threadIdx.x == 0) {
            volatile uchar gpu_cpu_read_idx = *dgpu_cpu_read_idx;
            volatile uchar gpu_cpu_write_idx = *dgpu_cpu_write_idx;
            __threadfence_system();
            if (was_deq && (gpu_cpu_read_idx != (gpu_cpu_write_idx + 1) % QUEUENODES)) {
                // Enqueue
                dev_gpu_cpu_queues[blockIdx.x].queue_array[gpu_cpu_write_idx].req_id = req_id;
                *dgpu_cpu_write_idx = (gpu_cpu_write_idx + 1) % QUEUENODES;
                __threadfence_system();
                req_count++;
                __threadfence();
                was_deq = false;
                //printf("GPUp: Req #%d was completed in TB #%d by thread %d, RC = %d\n", req_id, blockIdx.x, threadIdx.x, req_count);
            }
        }
        //printf("GPU: Req #%d in TB: %d by thread %d\n", req_count, blockIdx.x, threadIdx.x);
    }
}

void print_usage_and_die(char *progname) {
    printf("usage:\n");
    printf("%s streams <load (requests/sec)>\n", progname);
    printf("OR\n");
    printf("%s queue <#threads> <load (requests/sec)>\n", progname);
    exit(1);
}


enum {PROGRAM_MODE_STREAMS = 0, PROGRAM_MODE_QUEUE};
int main(int argc, char *argv[]) {

    int mode = -1;
    int threads_queue_mode = -1; /* valid only when mode = queue */
    double load = 0;
    if (argc < 3) print_usage_and_die(argv[0]);

    if        (!strcmp(argv[1], "streams")) {
        if (argc != 3) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_STREAMS;
        load = atof(argv[2]);
    } else if (!strcmp(argv[1], "queue")) {
        if (argc != 4) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_QUEUE;
        threads_queue_mode = atoi(argv[2]);
        load = atof(argv[3]);
    } else {
        print_usage_and_die(argv[0]);
    }

    uchar *images1; /* we concatenate all images in one huge array */
    uchar *images2;
    CUDA_CHECK( hipHostAlloc(&images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );
    CUDA_CHECK( hipHostAlloc(&images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );

    load_image_pairs(images1, images2);
    double t_start, t_finish;
    double total_distance;
#if 0
    /* using CPU */
    printf("\n=== CPU ===\n");
    int histogram1[256];
    int histogram2[256];
    t_start  = get_time_msec();
    for (int i = 0; i < NREQUESTS; i++) {
        int img_idx = i % N_IMG_PAIRS;
        image_to_histogram(&images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram1);
        image_to_histogram(&images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram2);
        total_distance += histogram_distance(histogram1, histogram2);
    }
    t_finish = get_time_msec();
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

    /* using GPU task-serial.. just to verify the GPU code makes sense */
    printf("\n=== GPU Task Serial ===\n");
    do {
        uchar *gpu_image1, *gpu_image2; // TODO: allocate with cudaMalloc
        int *gpu_hist1, *gpu_hist2; // TODO: allocate with cudaMalloc
        double *gpu_hist_distance; //TODO: allocate with cudaMalloc
        double cpu_hist_distance;
        cudaMalloc(&gpu_image1, IMG_DIMENSION * IMG_DIMENSION);
        cudaMalloc(&gpu_image2, IMG_DIMENSION * IMG_DIMENSION);
        cudaMalloc(&gpu_hist1, 256 * sizeof(int));
        cudaMalloc(&gpu_hist2, 256 * sizeof(int));
        cudaMalloc(&gpu_hist_distance, 256 * sizeof(double));

        total_distance = 0;
        t_start = get_time_msec();
        for (int i = 0; i < NREQUESTS; i++) {
            int img_idx = i % N_IMG_PAIRS;
            cudaMemcpy(gpu_image1, &images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, cudaMemcpyHostToDevice);
            cudaMemcpy(gpu_image2, &images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, cudaMemcpyHostToDevice);
            cudaMemset(gpu_hist1, 0, 256 * sizeof(int));
            cudaMemset(gpu_hist2, 0, 256 * sizeof(int));
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image1, gpu_hist1);
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image2, gpu_hist2);
            gpu_histogram_distance<<<1, 256>>>(gpu_hist1, gpu_hist2, gpu_hist_distance);
            cudaMemcpy(&cpu_hist_distance, gpu_hist_distance, sizeof(double), cudaMemcpyDeviceToHost);
            total_distance += cpu_hist_distance;
        }
        CUDA_CHECK(cudaDeviceSynchronize());
        t_finish = get_time_msec();
        printf("average distance between images %f\n", total_distance / NREQUESTS);
        printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);
        CUDA_CHECK( cudaFree(gpu_image1) );
        CUDA_CHECK( cudaFree(gpu_image2) );
        CUDA_CHECK( cudaFree(gpu_hist1) );
        CUDA_CHECK( cudaFree(gpu_hist2) );
        CUDA_CHECK( cudaFree(gpu_hist_distance) );
    } while (0);
#endif
    /* now for the client-server part */
    printf("\n=== Client-Server ===\n");
    total_distance = 0;
    double *req_t_start = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_start, 0, NREQUESTS * sizeof(double));

    double *req_t_end = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_end, 0, NREQUESTS * sizeof(double));

    struct rate_limit_t rate_limit;
    rate_limit_init(&rate_limit, load, 0);

    /* TODO allocate / initialize memory, streams, etc... */
    uchar *gpu_image1, *gpu_image2; // TODO: allocate with cudaMalloc
    int *gpu_hist1, *gpu_hist2; // TODO: allocate with cudaMalloc
    double *gpu_hist_distance; //TODO: allocate with cudaMalloc
    double *cpu_hist_distance;

    streamNode streams_array[NSTREAMS] = {0};
    int free_streams = NSTREAMS;

    double ti = get_time_msec();
    if (mode == PROGRAM_MODE_STREAMS) {
    	// Allocate CUDA memory for STREAMS
        CUDA_CHECK(hipMalloc(&gpu_image1, IMG_DIMENSION * IMG_DIMENSION * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_image2, IMG_DIMENSION * IMG_DIMENSION * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_hist1, 256 * sizeof(int) * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_hist2, 256 * sizeof(int) * NSTREAMS));
        CUDA_CHECK(hipMalloc(&gpu_hist_distance, 256 * sizeof(double) * NSTREAMS));
        CUDA_CHECK(hipHostAlloc(&cpu_hist_distance, sizeof(double), 0));

        // Init array of stream nodes
        for (int j = 0; j < NSTREAMS; j++) {
        	streams_array[j].stream_id = j;
        	streams_array[j].req_in_processing = -1;
        	CUDA_CHECK( hipStreamCreate(&streams_array[j].Stream));
        }

        for (int i = 0; i < NREQUESTS; i++) {

            /* TODO query (don't block) streams for any completed requests.
               update req_t_end of completed requests
               update total_distance */

            for (int j = 0; j < NSTREAMS; j++) {
                if ( streams_array[j].req_in_processing != -1) {
                    if ( hipStreamQuery(streams_array[j].Stream) == hipSuccess) {
                        req_t_end[streams_array[j].req_in_processing] = get_time_msec();
                        CUDA_CHECK( hipMemcpyAsync(cpu_hist_distance, &(gpu_hist_distance[streams_array[j].stream_id * 256]), sizeof(double), hipMemcpyDeviceToHost, streams_array[j].Stream));
                        total_distance += *cpu_hist_distance;
                        streams_array[j].req_in_processing = -1;
                        free_streams++;
                    }
                }
            }

            rate_limit_wait(&rate_limit);
            req_t_start[i] = get_time_msec();
            int img_idx = i % N_IMG_PAIRS;

            /* TODO place memcpy's and kernels in a stream */
            if (free_streams > 0) {

            	// Find first free stream
            	streamNode *busy_streams;
            	for (int j = 0; j < NSTREAMS; j++) {
            		if (streams_array[j].req_in_processing == -1) {
            			busy_streams = &streams_array[j];
            			break;
            		}
            	}

            	busy_streams->req_in_processing = i;
            	free_streams--;

                // Enqueue data copy and kernel execution for selected stream
                CUDA_CHECK(hipMemcpyAsync(&(gpu_image1[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice, busy_streams->Stream));
                CUDA_CHECK(hipMemcpyAsync(&(gpu_image2[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice, busy_streams->Stream));
                CUDA_CHECK(hipMemsetAsync(&(gpu_hist1[busy_streams->stream_id * 256]), 0, 256 * sizeof(int), busy_streams->Stream));
                CUDA_CHECK(hipMemsetAsync(&(gpu_hist2[busy_streams->stream_id * 256]), 0, 256 * sizeof(int), busy_streams->Stream));

                gpu_image_to_histogram<<<1, 1024, 0, busy_streams->Stream>>>(&(gpu_image1[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &(gpu_hist1[busy_streams->stream_id * 256]));
                gpu_image_to_histogram<<<1, 1024, 0, busy_streams->Stream>>>(&(gpu_image2[busy_streams->stream_id * IMG_DIMENSION * IMG_DIMENSION]), &(gpu_hist2[busy_streams->stream_id * 256]));
                gpu_histogram_distance<<<1, 256, 0, busy_streams->Stream>>>(&(gpu_hist1[busy_streams->stream_id * 256]), &(gpu_hist2[busy_streams->stream_id * 256]), &(gpu_hist_distance[busy_streams->stream_id * 256]));
            }
        }
        /* TODO now make sure to wait for all streams to finish */
    	for (int j = 0; j < NSTREAMS; j++) {
    		if (streams_array[j].req_in_processing != -1) {
    			CUDA_CHECK( hipStreamSynchronize(streams_array[j].Stream) );
    			req_t_end[streams_array[j].req_in_processing] = get_time_msec();
    		}
    	}

        for (int j = 0; j < NSTREAMS; j++) {
        	CUDA_CHECK( hipStreamDestroy(streams_array[j].Stream));
        }

        CUDA_CHECK( hipFree(gpu_hist_distance) );
        CUDA_CHECK( hipHostFree(cpu_hist_distance) );

    } else if (mode == PROGRAM_MODE_QUEUE) {
    	// Check for CUDA device and calculate amount of CPU<->GPU queues accordingly to it's capabilities
    	int deviceCount = 0, cuda_device = 0;
    	CUDA_CHECK( hipGetDeviceCount(&deviceCount) );
    	if (deviceCount > 0) {
    		printf("CUDA Device(s) found, will use first available device: ");
    	} else {
    		printf("No CUDA Device found, terminating the program!\n");
    		assert(0);
    	}
    	hipSetDevice(cuda_device);
    	hipDeviceProp_t deviceProp;
    	hipGetDeviceProperties(&deviceProp, cuda_device);
    	printf("%s\n", deviceProp.name);
    	if (	threads_queue_mode > IMG_DIMENSION * IMG_DIMENSION  ||
    			threads_queue_mode <= 0    ||
    			deviceProp.maxThreadsPerBlock < IMG_DIMENSION * IMG_DIMENSION
    		)
    	{
    		printf("Wrong amount of threads requested for 32x32 images or your device incapable to run 1024 threads in one block,\nPlease enter #threads = 1024 or less.\n");
    		assert (0);
    	}

    	if ( deviceProp.canMapHostMemory == 0){
    		printf("Your CUDA Device doesn't support cudaDeviceMapHost, terminating the program!\n");
    		assert(0);
    	}

    	unsigned int max_simult_blocks = deviceProp.multiProcessorCount * (deviceProp.maxThreadsPerMultiProcessor / threads_queue_mode);
    	printf("This device is capable to run %d thread blocks simultaneously.\n", max_simult_blocks);
    	if ( (deviceProp.regsPerBlock / (max_simult_blocks * threads_queue_mode)) < MAXREGCOUNT) {
    		max_simult_blocks = deviceProp.regsPerBlock / ( threads_queue_mode * MAXREGCOUNT );
    		printf("Amount of running blocks (queue pairs) was reduced to %d due to device Registers limitation\n", max_simult_blocks);
    	}
    	if ( (deviceProp.sharedMemPerBlock / (2 * (IMG_DIMENSION * IMG_DIMENSION) + 2 * sizeof (int) * 256 + sizeof (double) * 256)) < 1) {
    		printf("No enough Shared memory per block, terminating the program!\n");
    		assert(0);
    	}

    	// Create CPU<->GPU queues
    	volatile threads_queue *cpu_gpu_queues;
    	volatile threads_queue *gpu_cpu_queues;
    	CUDA_CHECK( hipHostAlloc(&gpu_cpu_queues, sizeof (threads_queue) * max_simult_blocks, hipHostMallocMapped) );
		CUDA_CHECK( hipHostAlloc(&cpu_gpu_queues, sizeof (threads_queue) * max_simult_blocks, hipHostMallocMapped) );

    	for (int i = 0; i < max_simult_blocks; i++) {
    		cpu_gpu_queues[i].read_index = 0;
    		cpu_gpu_queues[i].write_index = 0;
    		gpu_cpu_queues[i].read_index = 0;
    		gpu_cpu_queues[i].write_index = 0;
    	}


    	float *gpu_total_distance_f;
    	thread_node *cpu_hist_distance_node;

        CUDA_CHECK( hipMalloc(&gpu_image1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION) );
        CUDA_CHECK( hipMalloc(&gpu_image2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION) );
        CUDA_CHECK( hipMalloc(&gpu_hist1, sizeof (int) * 256 * N_IMG_PAIRS) );
        CUDA_CHECK( hipMalloc(&gpu_hist2, sizeof (int) * 256 * N_IMG_PAIRS) );
        CUDA_CHECK( hipMalloc(&gpu_total_distance_f, sizeof (float) * 256 * max_simult_blocks) );
        CUDA_CHECK( hipHostAlloc(&cpu_hist_distance_node, sizeof(thread_node), 0) );

        CUDA_CHECK( hipMemcpy(gpu_image1, images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice) );
        CUDA_CHECK( hipMemcpy(gpu_image2, images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice) );
        CUDA_CHECK( hipMemset(gpu_total_distance_f,0 , sizeof(float) * 256 * max_simult_blocks));

        volatile threads_queue *dev_cpu_gpu_queues, *dev_gpu_cpu_queues;
        CUDA_CHECK( hipHostGetDevicePointer ((void **)&dev_gpu_cpu_queues, (void *)gpu_cpu_queues, 0) );
        CUDA_CHECK( hipHostGetDevicePointer ((void **)&dev_cpu_gpu_queues, (void *)cpu_gpu_queues, 0) );

        //Start CUDA kernel
        process_queues<<<max_simult_blocks, threads_queue_mode>>>(dev_gpu_cpu_queues, dev_cpu_gpu_queues, max_simult_blocks);

        for (unsigned int i = 0, k = 0; k < NREQUESTS ;) {

            /* TODO check producer consumer queue for any responses.
               don't block. if no responses are there we'll check again in the next iteration
               update req_t_end of completed requests
               update total_distance */
        	for (int j = 0; j < max_simult_blocks; j++) {
        		// Get current GPU->CPU queue indexes
                volatile uchar *pgpu_cpu_read_index = &gpu_cpu_queues[j].read_index;
                volatile uchar *pgpu_cpu_write_index = &gpu_cpu_queues[j].write_index;
                volatile uchar read_idx  = *pgpu_cpu_read_index;
                volatile uchar write_idx = *pgpu_cpu_write_index;
                __sync_synchronize();
                volatile int req_id = gpu_cpu_queues[j].queue_array[read_idx].req_id;
                __sync_synchronize();
        		// Dequeue completed
        		if (read_idx != write_idx) {
        			req_t_start[req_id] = get_time_msec();
                    *pgpu_cpu_read_index = (read_idx + 1) % QUEUENODES;
                    __sync_synchronize();
                    //printf("CPU: GPU-CPU read index #%d was updated by TB #%d, completion #%d\n", read_idx, j, k);
                    // Advance completed requests counter
                    k++;
                }
        	}


            if (i < NREQUESTS) {
            	rate_limit_wait(&rate_limit);
            	int queue_idx = i % max_simult_blocks;
            	req_t_start[i] = get_time_msec();
				volatile uchar read_idx = cpu_gpu_queues[queue_idx].read_index;
				volatile uchar write_idx = cpu_gpu_queues[queue_idx].write_index;
				volatile uchar *pcpu_gpu_write_index = &cpu_gpu_queues[queue_idx].write_index;
				__sync_synchronize();
				if (read_idx != (write_idx + 1) % QUEUENODES) {
					// Enqueue
					cpu_gpu_queues[queue_idx].queue_array[write_idx].req_id = i;
					*pcpu_gpu_write_index = (write_idx + 1) % QUEUENODES;
					__sync_synchronize();
					//printf("CPU: CPU-GPU write index #%d was increased by thread %d\n", write_idx, queue_idx);
					// Advance request id
					i++;
				}
            }
        }
        /* TODO wait until you have responses for all requests */

        // Release memory allocations specific for threads flow
        CUDA_CHECK( hipHostFree((void *)gpu_cpu_queues) );
        CUDA_CHECK( hipHostFree((void *)cpu_gpu_queues) );
        CUDA_CHECK( hipFree(gpu_total_distance_f) );
        CUDA_CHECK( hipHostFree(cpu_hist_distance_node) );
    } else {
        assert(0);
    }
    double tf = get_time_msec();

    CUDA_CHECK( hipFree(gpu_image1) );
    CUDA_CHECK( hipFree(gpu_image2) );
    CUDA_CHECK( hipFree(gpu_hist1) );
    CUDA_CHECK( hipFree(gpu_hist2) );

    double avg_latency = 0;
    for (int j = 0; j < NREQUESTS; j++) {
        avg_latency += (req_t_end[j] - req_t_start[j]);
    }
    avg_latency /= NREQUESTS;

    printf("mode = %s\n", mode == PROGRAM_MODE_STREAMS ? "streams" : "queue");
    printf("load = %lf (req/sec)\n", load);
    if (mode == PROGRAM_MODE_QUEUE) printf("threads = %d\n", threads_queue_mode);
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (tf - ti) * 1e+3);
    printf("average latency = %lf (msec)\n", avg_latency);

    return 0;
}
